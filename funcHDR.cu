#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <iomanip>

#define REDUCTION_THREADS 128
#define DEBUG 0 //Activar la impresi�n de resultados por consola
#define THRUST 0 //Activar el uso de la librer�a Thrust en lugar de la implementaci�n en CUDA
#define SHARED 0 //Usar memoria compartida en la implementaci�n de b�squeda de m�nimos y m�ximos

#if THRUST == 1
//Thrust libs
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/binary_search.h>

// simple routine to print contents of a vector
template <typename Vector>
void print_vector(const std::string& name, const Vector& v)
{
  typedef typename Vector::value_type T;
  std::cout << "  " << std::setw(20) << name << "  ";
  thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
  std::cout << std::endl;
}
#endif


#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__ void parallelMinMax(float* min, float* max, int len, int threads)
{
  int threadID = threadIdx.x + blockDim.x * blockIdx.x;

  float tempMin = min[threadID + threads];
  min[threadID] = tempMin <= min[threadID] ? tempMin : min[threadID];

  float tempMax = max[threadID + threads];
  max[threadID] = tempMax >= max[threadID] ? tempMax : max[threadID];

}

__global__ void parallelMinMaxInit(const float* input, float* min, float* max, int len, int threads)
{
  int threadID = threadIdx.x + blockDim.x * blockIdx.x;

  float tempMin = (threadID + threads) < len ? input[threadID + threads] : input[threadID];
  min[threadID] = tempMin <= input[threadID] ? tempMin : input[threadID];

  float tempMax = tempMin;
  max[threadID] = tempMax >= input[threadID] ? tempMax : input[threadID];
}

__global__ void initSharedMinMax(const float* input, float* vmin, float* vmax,int len)
{
	__shared__ float sharedMax[REDUCTION_THREADS * 2];
	__shared__ float sharedMin[REDUCTION_THREADS * 2];

	int ThreadID = threadIdx.x + blockDim.x * 2 * blockIdx.x;
	// Rellamos los valores de los threads
	if (ThreadID > len)
	{
		sharedMax[threadIdx.x] = input[0] ;
		sharedMin[threadIdx.x] = input[0] ;

	} else
	{
		sharedMax[threadIdx.x] = input[ThreadID];
		sharedMin[threadIdx.x] = sharedMax[threadIdx.x];

		// Rellenamos los valores desplazados.
		if (ThreadID + blockDim.x > len)
		{
			sharedMax[threadIdx.x + blockDim.x] = input[0];
			sharedMin[threadIdx.x + blockDim.x] = input[0];
		}
		else
		{
			sharedMax[threadIdx.x + blockDim.x] = input[ThreadID + blockDim.x];
			sharedMin[threadIdx.x + blockDim.x] = input[ThreadID + blockDim.x];
		}
	}

	__syncthreads();

	for (unsigned int desp = blockDim.x; desp > 0; desp /= 2)
	{
		if (threadIdx.x < desp) {
			sharedMax[threadIdx.x] = max(sharedMax[threadIdx.x], sharedMax[threadIdx.x + desp]);
			sharedMin[threadIdx.x] = min(sharedMin[threadIdx.x], sharedMin[threadIdx.x + desp]);
		}
		__syncthreads();
	}

	vmin[blockIdx.x] = sharedMin[0];
	vmax[blockIdx.x] = sharedMax[0];
}

__global__ void sharedMinMax(float* vmin, float* vmax, int len)
{
	__shared__ float sharedMax[REDUCTION_THREADS * 2];
	__shared__ float sharedMin[REDUCTION_THREADS * 2];

	int ThreadID = threadIdx.x + blockDim.x * 2 * blockIdx.x;
	// Rellamos los valores de los threads
	if (ThreadID > len)
	{
		sharedMax[threadIdx.x] = vmax[0];
		sharedMin[threadIdx.x] = vmin[0];

	}
	else
	{
		sharedMax[threadIdx.x] = vmax[ThreadID];
		sharedMin[threadIdx.x] = vmin[ThreadID];

		// Rellenamos los valores desplazados.
		if (ThreadID + blockDim.x > len)
		{
			sharedMax[threadIdx.x + blockDim.x] = vmax[0];
			sharedMin[threadIdx.x + blockDim.x] = vmin[0];
		}
		else
		{
			sharedMax[threadIdx.x + blockDim.x] = vmax[ThreadID + blockDim.x];
			sharedMin[threadIdx.x + blockDim.x] = vmin[ThreadID + blockDim.x];
		}
	}

	__syncthreads();

	for (unsigned int desp = blockDim.x; desp > 0; desp /= 2)
	{
		if (threadIdx.x < desp) {
			sharedMax[threadIdx.x] = max(sharedMax[threadIdx.x], sharedMax[threadIdx.x + desp]);
			sharedMin[threadIdx.x] = min(sharedMin[threadIdx.x], sharedMin[threadIdx.x + desp]);
		}
		__syncthreads();
	}
	vmin[blockIdx.x] = sharedMin[0];
	vmax[blockIdx.x] = sharedMax[0];
}

__global__ void histogram(const float* input, const size_t numCols, const size_t numRows, const float min, const float range, const size_t numBins, unsigned int* histogram)
{
  int threadID = threadIdx.x + blockDim.x * blockIdx.x;
  if (threadID > numRows)
    return;
  unsigned int maxID = threadID * numCols + numCols;
  for(unsigned int i = threadID * numCols; i < maxID; i++)
  {
    int bin = (input[i] - min) / range * numBins;
    //�ltimo bin no inclusivo al quedar 1*numBins, reajustamos con un min(bin, numBins-1)
    bin = bin >= numBins ? numBins - 1 : bin;
    atomicAdd(&(histogram[bin]), 1);
  }
}

__global__ void scanReduce(unsigned int* scan, int start, int offset, int threads, int len)
{
  int threadID = threadIdx.x + blockDim.x * blockIdx.x;
  int index = start + threadID * offset * 2;
  int index2 = index + offset;
  //if (index2 < len)
    scan[index2] += scan[index];
}

__global__ void scanReverse(unsigned int* scan, int start, int offset, int threads, int len)
{
  int threadID = threadIdx.x + blockDim.x * blockIdx.x;
  int index = start + threadID * offset * 2;
  int index2 = index + offset;

  float tmp = scan[index2];
  //if(index2 < len)
    scan[index2] += scan[index];
    scan[index] = tmp;

  /*if(threadID == 0)
  {
    scan[index] = 0;
  }*/
}

void getMaxMin(const float* input, int len, float&min, float&max, float& range)
{
	float* d_tempMin;
	float* d_tempMax;
	int numBlocks = (len + 1) / (REDUCTION_THREADS * 2) + 1;
	hipMalloc(&d_tempMin, numBlocks * sizeof(float));
	hipMalloc(&d_tempMax, numBlocks * sizeof(float));
	initSharedMinMax<< <numBlocks, REDUCTION_THREADS >> > (input, d_tempMin,d_tempMax, len);
	hipDeviceSynchronize();

	while (numBlocks > 1) {
		int numBlocksTemp = numBlocks;
		numBlocks = (numBlocks + 1) / (REDUCTION_THREADS * 2) + 1;
		sharedMinMax<< <numBlocks, REDUCTION_THREADS >> > (d_tempMin, d_tempMax, numBlocksTemp);
		hipDeviceSynchronize();
	}

	hipMemcpy(&min, d_tempMin, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&max, d_tempMax, sizeof(float), hipMemcpyDeviceToHost);
	range = max - min;
	hipFree(d_tempMin);
	hipFree(d_tempMax);
}

void calculate_cdf(const float* const d_logLuminance,
	unsigned int* const d_cdf,
	float &min_logLum,
	float &max_logLum,
	const size_t numRows,
	const size_t numCols,
	const size_t numBins)
{
	/* TODO
	  1) Encontrar el valor m�ximo y m�nimo de luminancia en min_logLum and max_logLum a partir del canal logLuminance
	  2) Obtener el rango a representar
	  3) Generar un histograma de todos los valores del canal logLuminance usando la formula
	  bin = (Lum [i] - lumMin) / lumRange * numBins
	  4) Realizar un exclusive scan en el histograma para obtener la distribuci�n acumulada (cdf)
	  de los valores de luminancia. Se debe almacenar en el puntero c_cdf
	*/

  unsigned int numThreads, numBlocks;

#if THRUST == 0

#if SHARED == 1
  float min, max, range2;
  getMaxMin(d_logLuminance, numRows * numCols, min, max, range2);
  min_logLum = min;
  max_logLum = max;
#else
#define REDUCTION_THREADS 32
  // Obtener m�ximo y m�nimo
  numThreads = (numRows * numCols) / 2;
  if (numThreads % 2 != 0) numThreads++;
  float* d_minArray, *d_maxArray;
  checkCudaErrors(hipMalloc(&d_minArray, numThreads * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_maxArray, numThreads * sizeof(float)));
  numBlocks = (numThreads - 1) / REDUCTION_THREADS + 1;
  parallelMinMaxInit << < numBlocks, REDUCTION_THREADS >> > (d_logLuminance, d_minArray, d_maxArray, numRows * numCols, numThreads);
  while (numThreads > 1) {
    numThreads = std::ceil((float)numThreads / 2.0f);
    numBlocks = (numThreads - 1) / REDUCTION_THREADS + 1;
    parallelMinMax << < numBlocks, REDUCTION_THREADS >> > (d_minArray, d_maxArray, numRows * numCols, numThreads);
    hipDeviceSynchronize();
  }
  checkCudaErrors(hipMemcpy(&min_logLum, d_minArray, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_maxArray, sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_minArray));
  checkCudaErrors(hipFree(d_maxArray));

#endif
  //Imprimir resultados de m�ximo, m�nimo y rango
  float range = max_logLum - min_logLum;
#if DEBUG == 1
  std::cout << "Min: " << min_logLum << " | Max: " << max_logLum << std::endl;
  std::cout << "Rango a representar: " << range << std::endl;
#endif //DEBUG

  //Obtener histograma
  numThreads = numRows;
  numBlocks = (numThreads - 1) / REDUCTION_THREADS + 1;
  histogram <<< dim3{ numBlocks, 1, 1 }, dim3{ REDUCTION_THREADS, 1, 1 } >>> (d_logLuminance, numCols, numRows, min_logLum, range, numBins, d_cdf);
  hipDeviceSynchronize();

  //Imprimir resultados del histograma
#if DEBUG == 1
  unsigned int* h_histogram = (unsigned int*)malloc(numBins * sizeof(unsigned int));
  checkCudaErrors(hipMemcpy(h_histogram, d_cdf, numBins * sizeof(unsigned int), hipMemcpyDeviceToHost));
  std::cout << "Histograma: ";
  unsigned int total = 0;
  for(int i = 0 ; i < numBins; i++)
  {
    if (i % 25 == 0)
      std::cout << std::endl;
    std::cout << h_histogram[i] << " ";
    total += h_histogram[i];
  }
  std::cout << std::endl;
  std::cout << "Total: " << total << std::endl;
  std::cout << "Total num*col: " << numCols*numRows << std::endl;
  std::cout << "Numbins: " << numBins << std::endl;
#endif //DEBUG

  //Calcular exclusive scan a partir del histograma
  unsigned int len = numBins;
  numThreads = len / 2;
  int d = 0;
  unsigned int start, offset = 1;
  do {
    start = offset - 1;
    numBlocks = (numThreads - 1) / REDUCTION_THREADS + 1;
    scanReduce << < numBlocks, REDUCTION_THREADS >> > (d_cdf, start, offset, numThreads, len);
    d++;
    numThreads >>= 1;
    hipDeviceSynchronize();
  } while ((offset <<= 1) <= len/2);
  checkCudaErrors(hipMemset(&d_cdf[len-1], 0, sizeof(unsigned int)));
  numThreads = 1;
  while (d > 0)
  {
    d--;
    offset = (1 << d);
    start = offset - 1;
    numBlocks = (numThreads - 1) / REDUCTION_THREADS + 1;
    scanReverse << < numBlocks, REDUCTION_THREADS >> > (d_cdf, start, offset, numThreads, len);
    numThreads <<= 1;
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();

  //Imprimir resultados del exclusive scan
#if DEBUG == 1
  unsigned int* h_scan = (unsigned int*)malloc(numBins * sizeof(unsigned int));
  checkCudaErrors(hipMemcpy(h_scan, d_cdf, numBins * sizeof(unsigned int), hipMemcpyDeviceToHost));
  std::cout << "Exclusive scan: ";
  for(int i = 0; i < numBins; i++)
  {
    if (i % 25 == 0)
      std::cout << std::endl;
    std::cout << h_scan[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "Esperado: ";
  total = 0;
  for (int i = 0; i < numBins; i++)
  {
    if (i % 25 == 0)
      std::cout << std::endl;
    std::cout << total << " ";
    total += h_histogram[i];
  }
  std::cout << std::endl;
  free(h_histogram);
#endif //DEBUG

#else // Usar THRUST

  //Calcular m�nimo, m�ximo y range con THRUST
  unsigned int len = numRows * numCols;
  thrust::device_ptr<const float> d_t_luminancePtr = thrust::device_pointer_cast(d_logLuminance);
  thrust::device_vector<float> d_t_luminance(d_t_luminancePtr, d_t_luminancePtr + len);
  auto minmax = thrust::minmax_element(d_t_luminance.begin(), d_t_luminance.end());
  min_logLum = *minmax.first;
  max_logLum = *minmax.second;
  float range = max_logLum - min_logLum;
#if DEBUG == 1
  std::cout << "Min: " << min_logLum << " | Max: " << max_logLum << std::endl;
  std::cout << "Rango a representar: " << range << std::endl;
#endif //DEBUG
  //Histograma adaptado de: https://github.com/thrust/thrust/blob/master/examples/histogram.cu
  /*
   *thrust::device_vector<int> histogram;
  thrust::sort(d_t_luminance.begin(), d_t_luminance.end());
  histogram.resize(numBins);
  thrust::counting_iterator<int> search_begin(0);
  thrust::upper_bound(d_t_luminance.begin(), d_t_luminance.end(),
    search_begin, search_begin + numBins,
    histogram.begin());
  print_vector("cumulative histogram", histogram);
  thrust::adjacent_difference(histogram.begin(), histogram.end(),
    histogram.begin());
#if DEBUG == 1
  std::cout << "Histograma: ";
  thrust::copy(histogram.begin(), histogram.end(), std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;
#endif //DEBUG
  */

  //Histograma de ejemplos de Thrust no funciona
  hipDeviceSynchronize();
  unsigned int numThreads = numRows;
  unsigned int numBlocks = (numThreads - 1) / REDUCTION_THREADS + 1;
  histogram << < dim3{ numBlocks, 1, 1 }, dim3{ REDUCTION_THREADS, 1, 1 } >> > (d_logLuminance, numCols, numRows, min_logLum, range, numBins, d_cdf);
  hipDeviceSynchronize();

  //Imprimir resultados de histograma
#if DEBUG == 1
  unsigned int* h_histogram = (unsigned int*)malloc(numBins * sizeof(unsigned int));
  checkCudaErrors(hipMemcpy(h_histogram, d_cdf, numBins * sizeof(unsigned int), hipMemcpyDeviceToHost));
  std::cout << "Histograma: ";
  unsigned int total = 0;
  for (int i = 0; i < numBins; i++)
  {
    if (i % 25 == 0)
      std::cout << std::endl;
    std::cout << h_histogram[i] << " ";
    total += h_histogram[i];
  }
  std::cout << std::endl;
  std::cout << "Total: " << total << std::endl;
  std::cout << "Total num*col: " << numCols * numRows << std::endl;
  std::cout << "Numbins: " << numBins << std::endl;
  free(h_histogram);
#endif //DEBUG  


  thrust::device_ptr<unsigned int> d_t_histogramPtr = thrust::device_pointer_cast(d_cdf);
  thrust::device_vector<unsigned int> d_t_histogram(d_t_histogramPtr, d_t_histogramPtr + numBins);
  thrust::device_ptr<unsigned int> d_t_cdf = thrust::device_pointer_cast(d_cdf);
  thrust::exclusive_scan(d_t_histogram.begin(), d_t_histogram.end(), d_t_cdf);
#if DEBUG == 1
  std::cout << "Exclusive scan: ";
  for (int i = 0; i < numBins; i++)
  {
    if (i % 25 == 0)
      std::cout << std::endl;
    std::cout << d_t_cdf[i] << " ";
  }
  std::cout << std::endl;
#endif //DEBUG
#endif //THRUST
}
